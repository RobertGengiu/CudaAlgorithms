﻿#include <iostream>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>;﻿

using namespace std;

__global__ void AddIntsCuda(int *a, int *b)
{
	int i = threadIdx.x;
	a[i] += b[i];
}

__global__ void InterChangeCuda(int *a, int *b)
{
	 int i = threadIdx.x;
	int temp = a[i];
	a[i] = b[i];
	b[i] = a[i];
}


int main() {
	int a = 5, b = 9;
	int *d_a, *d_b;

	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	AddIntsCuda<<< 1, 1 >>>(d_a, d_b);

	hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

	cout << "The result is: " << a << endl;

	hipFree(d_a);
	hipFree(d_b);

	return 0;
}

